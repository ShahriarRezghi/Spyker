#include "base.cuh"

namespace Spyker
{
namespace Core
{
namespace CUDA
{
bool LightConv = false;

#ifdef SPYKER_USE_CUDNN
struct Workspace
{
    Size size = 0;
    void *ptr = nullptr;

    ~Workspace()
    {
        if (ptr != nullptr) cuda_dealloc(ptr);
    }
} space;

struct Conv
{
    hipdnnTensorDescriptor_t input;
    hipdnnTensorDescriptor_t output;
    hipdnnFilterDescriptor_t kernel;
    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnConvolutionDescriptor_t conv;

    Len4 _input;
    Len4 _kernel;
    Len4 _output;
    Len2 _stride;
    Len2 _pad;

    Conv(Len4 _input, Len4 _kernel, Len4 _output, Len2 _stride, Len2 _pad)
        : _input(_input), _kernel(_kernel), _output(_output), _stride(_stride), _pad(_pad)
    {
        if (!cudnn_static) cudnn_static = std::unique_ptr<cudnn>(new cudnn);

        CudnnCheck(hipdnnCreateTensorDescriptor(&input));
        CudnnCheck(hipdnnCreateTensorDescriptor(&output));
        CudnnCheck(hipdnnCreateFilterDescriptor(&kernel));
        CudnnCheck(hipdnnCreateConvolutionDescriptor(&conv));

        CudnnCheck(hipdnnSetConvolution2dDescriptor(conv, _pad.y, _pad.x, _stride.y, _stride.x, 1, 1,  //
                                                   HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
        CudnnCheck(hipdnnSetTensor4dDescriptor(input, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,  //
                                              _input.t, _input.z, _input.y, _input.x));
        CudnnCheck(hipdnnSetFilter4dDescriptor(kernel, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,  //
                                              _kernel.t, _kernel.z, _kernel.y, _kernel.x));
        CudnnCheck(hipdnnSetTensor4dDescriptor(output, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,  //
                                              _output.t, _output.z, _output.y, _output.x));

        int index = 0, count = HIPDNN_CONVOLUTION_FWD_ALGO_COUNT, returned;
        hipdnnConvolutionFwdAlgoPerf_t algos[HIPDNN_CONVOLUTION_FWD_ALGO_COUNT];
        if (LightConv)
        {
#if SPYKER_CUDNN_VERSION >= 7
            CudnnCheck(cudnnGetConvolutionForwardAlgorithm_v7(  //
                cudnn_static->handle, input, kernel, conv, output, count, &returned, algos));
#else
            CudnnCheck(hipdnnFindConvolutionForwardAlgorithm(  //
                cudnn_static->handle, input, kernel, conv, output, count, &returned, algos));
#endif

            for (Size i = 0; i < returned; ++i)
                if (algos[i].algo == HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM ||          //
                    algos[i].algo == HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM ||  //
                    algos[i].algo == HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT ||                 //
                    algos[i].algo == HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD)
                {
                    index = i;
                    break;
                }
        }
        else
        {
            CudnnCheck(hipdnnFindConvolutionForwardAlgorithm(  //
                cudnn_static->handle, input, kernel, conv, output, count, &returned, algos));
        }

#if SPYKER_CUDNN_VERSION >= 7
        CudnnCheck(hipdnnSetConvolutionMathType(conv, algos[index].mathType));
#endif

        algo = algos[index].algo;
        if (algos[index].memory != 0 && space.size < algos[index].memory)
        {
            space.size = algos[index].memory;
            if (space.ptr != nullptr) cuda_dealloc(space.ptr);
            space.ptr = cuda_alloc(space.size);
        }
    }
    ~Conv()
    {
        hipdnnDestroyTensorDescriptor(input);
        hipdnnDestroyFilterDescriptor(kernel);
        hipdnnDestroyTensorDescriptor(output);
        hipdnnDestroyConvolutionDescriptor(conv);
    }
    void operator()(F32 *input_, F32 *kernel_, F32 *output_)
    {
        F32 alpha = 1, beta = 0;
        CudnnCheck(hipdnnConvolutionForward(cudnn_static->handle, &alpha, input, input_, kernel, kernel_,  //
                                           conv, algo, space.ptr, space.size, &beta, output, output_));
    }
    bool comp(Len4 input_, Len4 kernel_, Len4 output_, Len2 stride_, Len2 pad_)
    {
        return _input == input_ && _kernel == kernel_ && _output == output_ && _stride == stride_ && _pad == pad_;
    }
};

std::vector<std::shared_ptr<Conv>> conv_handle;

void conv_clear()
{
    conv_handle.clear();
    if (space.ptr != nullptr) cuda_dealloc(space.ptr);
    space.size = 0, space.ptr = nullptr;
}

Conv &conv_find(Len4 input, Len4 kernel, Len4 output, Len2 stride, Len4 pad)
{
    auto pad_ = (pad.t != pad.y || pad.z != pad.x) ? Len2{0, 0} : Len2{pad.t, pad.z};
    for (auto conv : conv_handle)
        if (conv->comp(input, kernel, output, stride, pad_)) return *conv.get();
    conv_handle.push_back(std::shared_ptr<Conv>(new Conv(input, kernel, output, stride, pad_)));
    return *conv_handle.back().get();
}

void conv_(Vec4<F32> input, Vec4<F32> kernel, Vec4<F32> output, Len2 stride, Len4 pad)
{
    Conv &conv = conv_find(input.len(), kernel.len(), output.len(), stride, pad);
    conv(input.data, kernel.data, output.data);
}

void light_conv(bool light)
{
    SpykerCompare(space.ptr, ==, (void *)nullptr, "CUDA:Conv",
                  "Light convolution can't be set after using convolutional layers.");
    LightConv = light;
}

#else
void conv_(Vec4<F32> input, Vec4<F32> kernel, Vec4<F32> output, Len2 stride, Len4 pad)
{
    SpykerAssert(false, "CPU::Conv", "Conv operation needs cuDNN to work.");
}

void conv_clear() {}

void light_conv(bool light) { LightConv = light; }
#endif

template <typename T>
void conv_(Vec4<T> input, Vec4<F32> kernel, Vec4<F32> output, Len2 stride, Len4 pad)
{
    auto temp = init<F32>(input.t, input.z, input.y, input.x);
    copy(input, temp);
    conv_(temp, kernel, output, stride, pad);
    deinit(temp);
}

template <typename T>
void conv(Vec4<T> input_, Vec4<F32> kernel, Vec4<F32> output, Len2 stride, Len4 pad)
{
    auto input = input_;
    if ((pad.t != pad.y || pad.z != pad.x) && (pad.t != 0 || pad.z != 0 || pad.y != 0 || pad.x != 0))
    {
        input = init<T>(input_.t, input_.z, input_.y + pad.t + pad.y, input_.x + pad.z + pad.x);
        cuda_pad(todyn(Vec3<T>(input_.data, input_.t * input_.z, input_.y, input_.x)),
                 todyn(Vec3<T>(input.data, input.t * input.z, input.y, input.x)), pad, T(0));
    }
    conv_(input, kernel, output, stride, pad);
    if (input.data != input_.data) deinit(input);
}
}  // namespace CUDA

void cuda_conv(Dyn4 input, Dyn4 kernel, Dyn4 output, Len2 stride, Len4 pad)
{
    IfType(T, input.type, CUDA::conv<T>(input, kernel, output, stride, pad));
}
void cuda_conv_clear() { CUDA::conv_clear(); }
void cuda_light_conv(bool light) { CUDA::light_conv(light); }
}  // namespace Core
}  // namespace Spyker
