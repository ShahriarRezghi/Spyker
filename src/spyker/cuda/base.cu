#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "base.cuh"

namespace Spyker
{
namespace Core
{
namespace CUDA
{
void sync()
{
    CudaCheck(hipGetLastError());
    CudaCheck(hipDeviceSynchronize());
}

const char *curandGetErrorString(hiprandStatus_t error)
{
    if (error == HIPRAND_STATUS_SUCCESS) return "HIPRAND_STATUS_SUCCESS";
    if (error == HIPRAND_STATUS_VERSION_MISMATCH) return "HIPRAND_STATUS_VERSION_MISMATCH";
    if (error == HIPRAND_STATUS_NOT_INITIALIZED) return "HIPRAND_STATUS_NOT_INITIALIZED";
    if (error == HIPRAND_STATUS_ALLOCATION_FAILED) return "HIPRAND_STATUS_ALLOCATION_FAILED";
    if (error == HIPRAND_STATUS_TYPE_ERROR) return "HIPRAND_STATUS_TYPE_ERROR";
    if (error == HIPRAND_STATUS_OUT_OF_RANGE) return "HIPRAND_STATUS_OUT_OF_RANGE";
    if (error == HIPRAND_STATUS_LENGTH_NOT_MULTIPLE) return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
    if (error == HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED) return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
    if (error == HIPRAND_STATUS_LAUNCH_FAILURE) return "HIPRAND_STATUS_LAUNCH_FAILURE";
    if (error == HIPRAND_STATUS_PREEXISTING_FAILURE) return "HIPRAND_STATUS_PREEXISTING_FAILURE";
    if (error == HIPRAND_STATUS_INITIALIZATION_FAILED) return "HIPRAND_STATUS_INITIALIZATION_FAILED";
    if (error == HIPRAND_STATUS_ARCH_MISMATCH) return "HIPRAND_STATUS_ARCH_MISMATCH";
    if (error == HIPRAND_STATUS_INTERNAL_ERROR) return "HIPRAND_STATUS_INTERNAL_ERROR";
    return "Unknown";
}

const char *cublasGetErrorString(hipblasStatus_t error)
{
    if (error == HIPBLAS_STATUS_SUCCESS) return "HIPBLAS_STATUS_SUCCESS";
    if (error == HIPBLAS_STATUS_NOT_INITIALIZED) return "HIPBLAS_STATUS_NOT_INITIALIZED";
    if (error == HIPBLAS_STATUS_ALLOC_FAILED) return "HIPBLAS_STATUS_ALLOC_FAILED";
    if (error == HIPBLAS_STATUS_INVALID_VALUE) return "HIPBLAS_STATUS_INVALID_VALUE";
    if (error == HIPBLAS_STATUS_ARCH_MISMATCH) return "HIPBLAS_STATUS_ARCH_MISMATCH";
    if (error == HIPBLAS_STATUS_MAPPING_ERROR) return "HIPBLAS_STATUS_MAPPING_ERROR";
    if (error == HIPBLAS_STATUS_EXECUTION_FAILED) return "HIPBLAS_STATUS_EXECUTION_FAILED";
    if (error == HIPBLAS_STATUS_INTERNAL_ERROR) return "HIPBLAS_STATUS_INTERNAL_ERROR";
    if (error == HIPBLAS_STATUS_NOT_SUPPORTED) return "HIPBLAS_STATUS_NOT_SUPPORTED";
    if (error == HIPBLAS_STATUS_UNKNOWN) return "HIPBLAS_STATUS_UNKNOWN";
    return "Unknown";
}

std::unique_ptr<cublas> cublas_static;

#ifdef SPYKER_USE_CUDNN
std::unique_ptr<cudnn> cudnn_static;
#endif

template <typename T>
__device__ T maxv(T first, T second)
{
    return max(first, second);
}
__device__ C16 maxv(C16 first, C16 second) { return C16(max(F32(first), F32(second))); }

template <typename T>
__global__ void maxval(Cize isize, Cize osize, PTR(T, input), PTR(T, output))
{
    input += blockIdx.y * isize, output += blockIdx.y * osize;
    __shared__ T temp[Thread1D];
    temp[threadIdx.x] = Limits<T>::min();

    Cize idx = Index1D(T), end = min(isize, idx + Block1D(T));
    for (Cize i = idx; i < end; i += Thread1D) temp[threadIdx.x] = maxv(temp[threadIdx.x], input[i]);

    for (Cize i = Thread1D / 2; i > 0; i >>= 1)
    {
        __syncthreads();
        if (threadIdx.x < i) temp[threadIdx.x] = maxv(temp[threadIdx.x], temp[threadIdx.x + i]);
    }
    if (threadIdx.x == 0) output[blockIdx.x] = temp[0];
}

template <typename T>
Vec1<T> maxval_(Vec2<T> input, T *data)
{
    Vec2<T> max = {data, input.y, 0};
    while (true)
    {
        max.x = (input.x + Block1D(T) - 1) / Block1D(T);
        maxval<<<Config1D(T, 1, input.y, input.x)>>>(input.x, max.x, input.data, max.data);
        if (max.x == 1) return {max.data, max.y};
        input = max, max.data += max.size();
    }
}

template <typename T>
__device__ T minv(T first, T second)
{
    return min(first, second);
}
__device__ C16 minv(C16 first, C16 second) { return C16(min(F32(first), F32(second))); }

template <typename T>
__global__ void minval(Cize isize, Cize osize, PTR(T, input), PTR(T, output))
{
    input += blockIdx.y * isize, output += blockIdx.y * osize;
    __shared__ T temp[Thread1D];
    temp[threadIdx.x] = Limits<T>::max();

    Cize idx = Index1D(T), end = min(isize, idx + Block1D(T));
    for (Cize i = idx; i < end; i += Thread1D) temp[threadIdx.x] = minv(temp[threadIdx.x], input[i]);

    for (Cize i = Thread1D / 2; i > 0; i >>= 1)
    {
        __syncthreads();
        if (threadIdx.x < i) temp[threadIdx.x] = minv(temp[threadIdx.x], temp[threadIdx.x + i]);
    }
    if (threadIdx.x == 0) output[blockIdx.x] = temp[0];
}

template <typename T>
Vec1<T> minval_(Vec2<T> input, T *data)
{
    Vec2<T> max = {data, input.y, 0};
    while (true)
    {
        max.x = (input.x + Block1D(T) - 1) / Block1D(T);
        minval<<<Config1D(T, 1, input.y, input.x)>>>(input.x, max.x, input.data, max.data);
        if (max.x == 1) return {max.data, max.y};
        input = max, max.data += max.size();
    }
}

template <typename T>
__global__ void cuda_maxidx(Cize isize, Cize osize, PTR(U32, iindex), PTR(U32, oindex), PTR(T, ivalue), PTR(T, ovalue))
{
    if (iindex != nullptr) iindex += blockIdx.y * isize;
    oindex += blockIdx.y * osize;
    ivalue += blockIdx.y * isize;
    ovalue += blockIdx.y * osize;

    __shared__ U32 tindex[Thread1D];
    __shared__ T tvalue[Thread1D];

    tindex[threadIdx.x] = U32(-1), tvalue[threadIdx.x] = Limits<T>::min();
    Cize idx = Index1D(T), end = min(isize, idx + Block1D(T));
    for (Cize i = idx; i < end; i += Thread1D)
        if (ivalue[i] > tvalue[threadIdx.x])
        {
            tvalue[threadIdx.x] = ivalue[i];
            tindex[threadIdx.x] = (iindex == nullptr ? i : iindex[i]);
        }

    for (Cize i = Thread1D / 2; i > 0; i >>= 1)
    {
        __syncthreads();
        if (threadIdx.x < i)
            if (tvalue[threadIdx.x + i] > tvalue[threadIdx.x])
            {
                tvalue[threadIdx.x] = tvalue[threadIdx.x + i];
                tindex[threadIdx.x] = tindex[threadIdx.x + i];
            }
    }
    if (threadIdx.x == 0) oindex[blockIdx.x] = tindex[0], ovalue[blockIdx.x] = tvalue[0];
}

template <typename T>
Vec1<U32> maxidx_(Vec2<T> input_, U32 *index, T *data)
{
    T *ivalue = input_.data;
    Vec2<U32> max = {index, input_.y, 0};
    Vec2<U32> input = {nullptr, input_.y, input_.x};

    while (true)
    {
        max.x = (input.x + Block1D(T) - 1) / Block1D(T);
        cuda_maxidx<<<Config1D(T, 1, input.y, input.x)>>>(input.x, max.x, input.data, max.data, ivalue, data);
        if (max.x == 1) return {max.data, max.y};
        input = max, ivalue = data, max.data += max.size(), data += max.size();
    }
}

void *_maxval(Dyn2 input, void *data)  //
{
    IfType(T, input.type, return maxval_<T>(input, (T *)data).data);
}
void *_minval(Dyn2 input, void *data)  //
{
    IfType(T, input.type, return minval_<T>(input, (T *)data).data);
}
U32 *_maxidx(Dyn2 input, U32 *index, void *data)
{
    IfType(T, input.type, return maxidx_<T>(input, index, (T *)data).data);
}
}  // namespace CUDA
}  // namespace Core
}  // namespace Spyker
