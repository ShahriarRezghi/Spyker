#include "base.cuh"

namespace Spyker
{
namespace Core
{
namespace CUDA
{
void matmul_(F32 *A, F32 *B, F32 *C, bool AT, bool BT, int AR, int AC, int BR, int BC)
{
    if (!cublas_static) cublas_static = std::unique_ptr<cublas>(new cublas);

    F32 alpha = 1, beta = 0;
    int ATR = AT ? AC : AR, ATC = AT ? AR : AC, BTC = BT ? BR : BC;
    hipblasOperation_t AT_ = AT ? HIPBLAS_OP_T : HIPBLAS_OP_N, BT_ = BT ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasSgemm(cublas_static->handle, AT_, BT_, ATR, BTC, ATC, &alpha, A, AR, B, BR, &beta, C, ATR);
}

void matmul(F32 *A, F32 *B, F32 *C, bool AT, bool BT, int AR, int AC, int BR, int BC)
{
    matmul_(B, A, C, BT, AT, BC, BR, AC, AR);
}

void fc(Vec3<F32> input, Vec2<F32> kernel, Vec3<F32> output)
{
    matmul(input.data, kernel.data, output.data, false, true, input.z * input.y, input.x, kernel.y, kernel.x);
}

template <typename T>
void fc(Vec3<T> input, Vec2<F32> kernel, Vec3<F32> output)
{
    auto temp = init<F32>(input.z, input.y, input.x);
    copy(input, temp);
    fc(temp, kernel, output);
    deinit(temp);
}
}  // namespace CUDA

void cuda_fc(Dyn3 input, Dyn2 kernel, Dyn3 output) { IfType(T, input.type, CUDA::fc<T>(input, kernel, output)); }
}  // namespace Core
}  // namespace Spyker
